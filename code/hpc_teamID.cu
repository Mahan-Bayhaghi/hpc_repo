#include "hip/hip_runtime.h"
#include <pcap.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <jansson.h>
#include <netinet/ip.h>
#include <netinet/udp.h>
#include <netinet/ether.h>
#include <time.h>
#include <pthread.h>
#include <unistd.h>

#define MAX_PAYLOAD_SIZE 1500
#define NUMBER_OF_THREADS 3
#define THREAD_MEM_SIZE 300
#define CUDA_THREADS_PER_BLOCK 256

typedef struct {
    int id;
    char payload[MAX_PAYLOAD_SIZE];
} segment_t;

int segment_count = 0;
unsigned int udp_packet_count = 0;

void write_segments_to_json(segment_t *segments, int count, const char *filename) {
    json_t *json_segments = json_array();
    for (int i = 0; i < count; i++) {
        json_t *json_segment = json_object();
        json_object_set_new(json_segment, "id", json_integer(segments[i].id));
        json_object_set_new(json_segment, "payload", json_string(segments[i].payload));
        json_array_append_new(json_segments, json_segment);
    }
    json_dump_file(json_segments, filename, JSON_INDENT(4));
    json_decref(json_segments);
}

struct Arguments
{
    pcap_t* handle;
    segment_t** segments_ptr;
    int* mem_cnt;
    int thread_id;
} typedef Arguments;

pthread_mutex_t print_mutex = PTHREAD_MUTEX_INITIALIZER;
pthread_mutex_t next_mutex = PTHREAD_MUTEX_INITIALIZER;

__device__ unsigned short d_ntohs(unsigned short val) {
    return (val << 8) | (val >> 8);
}

__device__ int d_strncmp(const char *s1, const char *s2, size_t n) {
    while (n && *s1 && (*s1 == *s2)) {
        ++s1;
        ++s2;
        --n;
    }
    if (n == 0) {
        return 0;
    } else {
        return (*(unsigned char *)s1 - *(unsigned char *)s2);
    }
}

__device__ char* d_strchr(const char *s, int c) {
    while (*s != (char)c) {
        if (!*s++) {
            return 0;
        }
    }
    return (char *)s;
}

__device__ char* d_strncpy(char *dest, const char *src, size_t n) {
    char *ret = dest;
    while (n && (*dest++ = *src++)) {
        --n;
    }
    if (n) {
        while (--n) {
            *dest++ = 0;
        }
    }
    return ret;
}

__global__ void process_packets(const u_char *packet_data, segment_t *segments, int *mem_cnt, int total_packets) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_packets) return;

    const u_char *packet = packet_data + idx * MAX_PAYLOAD_SIZE;
    struct ether_header *eth_header = (struct ether_header *)packet;

    if (d_ntohs(eth_header->ether_type) == ETHERTYPE_IP) {
        struct ip *ip_header = (struct ip *)(packet + sizeof(struct ether_header));
        if (ip_header->ip_p == IPPROTO_UDP) {
            atomicAdd(&udp_packet_count, 1);

            struct udphdr *udp_header = (struct udphdr *)((u_char*)ip_header + sizeof(struct ip));
            char *payload = (char *)((u_char*)udp_header + sizeof(struct udphdr));
            int payload_len = d_ntohs(udp_header->uh_ulen) - sizeof(struct udphdr);

            for (int i = 0; i <= payload_len - 5; i++) {
                if (d_strncmp(payload + i, "SEG{", 4) == 0) {
                    char *start = payload + i + 4;
                    char *end = d_strchr(start, '}');
                    if (end) {
                        int seg_len = end - start;
                        if (seg_len > 0 && seg_len < MAX_PAYLOAD_SIZE) {
                            int mem_idx = atomicAdd(mem_cnt, 1);
                            d_strncpy(segments[mem_idx].payload, start, seg_len);
                            segments[mem_idx].payload[seg_len] = '\0';
                            break;
                        }
                    }
                }
            }
        }
    }
}

void* runner(void* args){
    printf("-Thread started\n");
    struct Arguments* arguments = (struct Arguments*) args;
    pcap_t* handle = arguments->handle;
    segment_t** segments_ptr = arguments->segments_ptr;
    int* mem_cnt = arguments->mem_cnt;
    printf("-Arguments received\n");
       
    int thread_id = arguments->thread_id;
    
    pthread_mutex_lock(&print_mutex);
    printf("thread with id %d initialized\n", thread_id);
    pthread_mutex_unlock(&print_mutex);

    struct pcap_pkthdr pkthdr;

    int flag = 1;
    while (flag) {

        pthread_mutex_lock(&next_mutex);
        u_char* packet = pcap_next(handle, &pkthdr);
        pthread_mutex_unlock(&next_mutex);
        if (packet == NULL){
            printf("no more packets available for thread with ID %d\n", thread_id);
            pthread_exit(NULL);
            return NULL;
        }

        pthread_mutex_lock(&print_mutex);
        // printf("thread with id %d captured next packet\n", thread_id);
        pthread_mutex_unlock(&print_mutex);

        // loaded and ready

        if (segments_ptr == NULL){
            pthread_mutex_lock(&print_mutex);
            printf("segment pointer was null and got reallocated\n");
            pthread_mutex_unlock(&print_mutex);
            segments_ptr = (segment_t**)malloc(sizeof(segment_t*));
        }

        // Allocate device memory
        u_char *d_packet_data;
        segment_t *d_segments;
        int *d_mem_cnt;

        int total_packets = 1; // This needs to be set appropriately
        hipMalloc((void**)&d_packet_data, MAX_PAYLOAD_SIZE * total_packets);
        hipMalloc((void**)&d_segments, sizeof(segment_t) * THREAD_MEM_SIZE);
        hipMalloc((void**)&d_mem_cnt, sizeof(int));

        // Copy data to device
        hipMemcpy(d_packet_data, packet, MAX_PAYLOAD_SIZE * total_packets, hipMemcpyHostToDevice);
        hipMemcpy(d_mem_cnt, mem_cnt, sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        int num_blocks = (total_packets + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
        process_packets<<<num_blocks, CUDA_THREADS_PER_BLOCK>>>(d_packet_data, d_segments, d_mem_cnt, total_packets);

        // Copy results back to host
        hipMemcpy(segments_ptr, d_segments, sizeof(segment_t) * THREAD_MEM_SIZE, hipMemcpyDeviceToHost);
        hipMemcpy(mem_cnt, d_mem_cnt, sizeof(int), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_packet_data);
        hipFree(d_segments);
        hipFree(d_mem_cnt);
    }

    return NULL;
}


int main(int argc, char *argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <pcap_file_path>\n", argv[0]);
        return 1;
    }

    clock_t start_time = clock();

    char *pcap_file = argv[1];
    char *json_file = "temp/output.json";
    pcap_t *handle;
    char errbuf[PCAP_ERRBUF_SIZE];
    // segment_t *segments = NULL;

    handle = pcap_open_offline(pcap_file, errbuf);

    if (handle == NULL) {
        fprintf(stderr, "Could not open adjusted pcap file %s: %s\n", pcap_file, errbuf);
        return 2;
    }

    struct Arguments *args_arr[NUMBER_OF_THREADS];
    
    pthread_t threads[NUMBER_OF_THREADS];
    int mem_cnt[NUMBER_OF_THREADS] = {0};
    segment_t segments[THREAD_MEM_SIZE * NUMBER_OF_THREADS];

    for (int i = 0; i < NUMBER_OF_THREADS; i++) {
        args_arr[i] = (struct Arguments*)malloc(sizeof(struct Arguments));
        args_arr[i]->handle = handle;
        args_arr[i]->mem_cnt = &mem_cnt[i];
        args_arr[i]->segments_ptr = &segments[i * THREAD_MEM_SIZE];
        args_arr[i]->thread_id = i;

        pthread_create(&threads[i], NULL, runner, (void*)args_arr[i]);
    }

    for (int i = 0; i < NUMBER_OF_THREADS; i++) {
        printf("Thread Output: %d\n", pthread_join(threads[i], NULL));
    }

    int total_segments = 0;
    for (int i = 0; i < NUMBER_OF_THREADS; i++) {
        total_segments += mem_cnt[i];
    }
    printf("Total SEGs captured: %d\n", total_segments);

    pcap_close(handle);

    write_segments_to_json(segments, total_segments, json_file);

    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    printf("Segments extracted and written to %s\n", json_file);
    printf("Total UDP packets: %d\n", udp_packet_count);
    printf("Elapsed time: %.2f seconds\n", elapsed_time);

    return 0;
}
